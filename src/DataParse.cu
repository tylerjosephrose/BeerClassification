#include "hip/hip_runtime.h"
#include <map>
#include <stdio.h>
#include <string>
#include <vector>

__constant__ char** c_tags[85];

int getBit(unsigned char *bytes, int bit) {
    return ((bytes[(bit/8)] >> (bit % 8)) & 1);
}

__device__ void setBit(unsigned char *bytes, int bit, int val) {
    if (val == 1)
        bytes[(bit/8)] |= (1 << (bit % 8));
    else
        bytes [(bit/8)] &= ~(1 << (bit % 8));
}

void printBits(unsigned char *ptr, int sizeInbytes) {
    for (int i = 0; i < sizeInBytes * 8; i++) {
        printf("%d", getBit(ptr, i));
    }
    printf("\n");
}

__global__ void description_to_tags(char **d_descs, unsigned char *d_results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    char* desc = d_descs[idx];
    
    for (char *tag = *c_tags; tag; tag=*++tags) {
        //Check if the description contains the tag
        int diff = tag
        while (*desc) {
            
        }
    }
}

void descriptionToTags(char** tags, char** rawData) {
    // make kernel call
    printf("in cuda code size: %d\n", rawData.size());
    for (std::map<std::string, std::vector<std::string> >::iterator it = rawData.begin(); it != rawData.end(); it++) {
        printf("%s: ", it->first.c_str());
        for (uint i = 0; i < it->second.size(); i++) {
            //printf("%s ", it->second[i].c_str());
        }
        printf("\n");
    }
    
    /* Since we have 11Gb of memory on my GPU we don't need to worry about memory...at 
    85*20 bytes for the tags, 2000 bytes per beer for description, 11 bytes per beer for results
    it would take around 5.5 million beers to run out of memory...We don't have that*/
    const dim3 blockSize(1024, 1, 1);
    const dim3 gridSize(ceil(rawData.size()/1024.0, 1, 1);
    
    char **d_descs;
    unsigned char *d_results, *parsedResults;
    
    hipMalloc(&d_descs, rawData.size()*2000);
    hipMalloc(&d_results, 11*rawData.size());
    cudaMemcpytoSymbol(c_tags, tags, tags.size()*20);
    hipMemcpy(d_descs, rawData, rawData.size()*2000, hipMemcpyHostToDevice);
    hipMemcpy(d_results, 0, 11*rawData.size(), hipMemcpyHostToDevice);
    
    description_to_tags<<<gridSize, blockSize>>>(d_descs, d_results);
    
    parsedResults = malloc(11*rawData.size());
    hipMemcpy(parsedResults, d_results, 11*rawData.size(), hipMemcpyDeviceToHost);
    hipFree(d_tags);
    hipFree(d_descs);
    hipFree(d_results);
}
