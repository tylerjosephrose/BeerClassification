#include "hip/hip_runtime.h"
#include <fstream>
#include <map>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <vector>

#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

int getBit(unsigned char *bytes, int bit) {
    return ((bytes[(bit/8)] >> (bit % 8)) & 1);
}

/*__device__ void setBit(unsigned char *bytes, int bit, int val) {
    if (val == 1)
        bytes[(bit/8)] |= (1 << (bit % 8));
    else
        bytes [(bit/8)] &= ~(1 << (bit % 8));
}

void printBits(unsigned char *ptr, int sizeInBytes) {
    for (int i = 0; i < sizeInBytes * 8; i++) {
        printf("%d", getBit(ptr, i));
    }
    printf("\n");
}*/

__global__ void description_to_tags(char **d_descs, unsigned char *d_results, int sizeEntries, char **d_tags, int sizeTags) {
    if (threadIdx.x > 0) 
        return;
    for (int num = 0; num < sizeTags; num++) {
        for (int i = 0; i < sizeEntries; i++) {
            int spacing = sizeEntries/sizeTags;
            int idx = (i + threadIdx.x*spacing) % sizeEntries;

            // Copy the desc locally so we don't have read conflicts
            int j = 0;
            char desc[2000];
            while (d_descs[idx][j] != '\0') {
                desc[j] = d_descs[idx][j];
                j++;
            }
            desc[j] = '\0';

            //char* tag = d_tags[threadIdx.x];
            char* tag = d_tags[num];
            int tagLength = 0;
            while (tag[tagLength] != '\0')
                tagLength++;
            
            int match = 0;
            bool positiveMatch = false;
            j = 0;
            while (desc[j] != '\0') {
                char descLetter = desc[j];
                if ('A'<=descLetter && descLetter<='Z'){
                    descLetter=char(((int)descLetter)+32);
                }

                if (descLetter == tag[match]) {
                    match++;
                    if (match == tagLength) {
                        positiveMatch = true;
                        break;
                    }
                }
                else
                    match = 0;
                j++;
            }
            
            if (positiveMatch) {
                //printf("Thread %d:%d is looking for %s\t%s\n", threadIdx.x, idx, tag, "True!");
                // Since 11 bytes are given for each entry we need to find the byte that we are in and add the specific flag we need
                switch (num % 8) {
                    case 0:
                        d_results[(num/8) * i] |= 0b10000000;
                        break;
                    case 1:
                        d_results[(num/8) * i] |= 0b01000000;
                        break;
                    case 2:
                        d_results[(num/8) * i] |= 0b00100000;
                        break;
                    case 3:
                        d_results[(num/8) * i] |= 0b00010000;
                        break;
                    case 4:
                        d_results[(num/8) * i] |= 0b00001000;
                        break;
                    case 5:
                        d_results[(num/8) * i] |= 0b00000100;
                        break;
                    case 6:
                        d_results[(num/8) * i] |= 0b00000010;
                        break;
                    case 7:
                        d_results[(num/8) * i] |= 0b00000001;
                        break;
                }
            }
            else {
                //printf("Thread %d:%d is looking for %s\t%s\n", threadIdx.x, idx, tag, "False");
                switch (num % 8) {
                    case 0:
                        d_results[(num/8) * i] &= ~0b10000000;
                        break;
                    case 1:
                        d_results[(num/8) * i] &= ~0b01000000;
                        break;
                    case 2:
                        d_results[(num/8) * i] &= ~0b00100000;
                        break;
                    case 3:
                        d_results[(num/8) * i] &= ~0b00010000;
                        break;
                    case 4:
                        d_results[(num/8) * i] &= ~0b00001000;
                        break;
                    case 5:
                        d_results[(num/8) * i] &= ~0b00000100;
                        break;
                    case 6:
                        d_results[(num/8) * i] &= ~0b00000010;
                        break;
                    case 7:
                        d_results[(num/8) * i] &= ~0b00000001;
                        break;
                }
            }
            __syncthreads();
        }
    }
}

std::map<std::string, std::vector<float> > dataConversion(std::map<std::string, std::vector<std::string> > rawData, std::vector<std::string> tags_internal) {
    const char** descData = (const char**) malloc(sizeof(char)*2000*rawData.size());
    // Collected data is other data that we need to return we collect it so we don't need to iterate through the map again and worry about order.
    // The data is the style of the beer, abv, ibu
    std::vector< std::vector<std::string> > collectedData;
    int i = 0;
    // Put desc data into char**
    for (std::map<std::string, std::vector<std::string> >::iterator it = rawData.begin(); it != rawData.end(); it++) {
        descData[i] = it->second[1].c_str();
        std::vector<std::string> data;
        data.push_back(it->second[4].c_str());
        data.push_back(it->second[2].c_str());
        data.push_back(it->second[3].c_str());
        collectedData.push_back(data);
        i++;
    }

    // Get Tag data
    const char* tags = (char*) malloc(sizeof(char) * 20 * tags_internal.size());
    memset(tags, '\0', 20*tags_internal.size());
    int i = 0;
    for (std::set<std::string>::iterator it = tags_internal.begin(); it != tags_internal.end(); ++it) {
        for (int j = 0; j < 20; j++) {
            tags[i*20+j] = (*it)[j];
        }
        i++;
    }

    std::map<std::string, std::vector<float> > results;
    
    /* Since we have 11Gb of memory on my GPU we don't need to worry about memory...at 
    85*20 bytes for the tags, 2000 bytes per beer for description, 11 bytes per beer for results
    it would take around 5.5 million beers to run out of memory...We don't have that*/
    const dim3 blockSize(tags_internal.size(), 1, 1);
    const dim3 gridSize(1, 1, 1);
    
    char **d_descs, **d_tags;
    unsigned char *d_results;
    
    unsigned char *parsedResults = (unsigned char*) malloc(11*rawData.size());
    CudaSafeCall(hipMalloc(&d_results, 11*rawData.size()));

    // Copy descs to device
    CudaSafeCall(hipMalloc(&d_descs, rawData.size()*sizeof(char*)));
    char **d_temp_desc;
    d_temp_desc = (char **)malloc(rawData.size()*sizeof(char *));
    for (int i = 0; i < rawData.size(); i++){
        CudaSafeCall(hipMalloc(&(d_temp_desc[i]), 2000*sizeof(char)));
        CudaSafeCall(hipMemcpy(d_temp_desc[i], descData[i], 2000*sizeof(char), hipMemcpyHostToDevice));
        CudaSafeCall(hipMemcpy(d_descs+i, &(d_temp_desc[i]), sizeof(char *), hipMemcpyHostToDevice));
    }
    free(d_temp_desc);

    // Copy tags to global memory
    CudaSafeCall(hipMalloc(&d_tags, tags_internal.size()*sizeof(char*)));

    char **d_temp_tags;
    d_temp_tags = (char **)malloc(tags_internal.size()*sizeof(char*));
    for (int i = 0; i < tags_internal.size(); i++) {
        CudaSafeCall(hipMalloc(&(d_temp_tags[i]), 20*sizeof(char)));
        CudaSafeCall(hipMemcpy(d_temp_tags[i], tags[i], 20*sizeof(char), hipMemcpyHostToDevice));
        CudaSafeCall(hipMemcpy(d_tags+i, &(d_temp_tags[i]), sizeof(char *), hipMemcpyHostToDevice));
    }
    free(d_temp_tags);
    
    // Copy tags to constant memory
    
    hipMemcpyToSymbol(HIP_SYMBOL(c_tags), tags, tags_internal.size()*20, 0, hipMemcpyHostToDevice);

    description_to_tags<<<gridSize, blockSize>>>(d_descs, d_results, rawData.size(), d_tags, tags_internal.size());

    CudaSafeCall(hipMemcpy(parsedResults, d_results, 11*rawData.size(), hipMemcpyDeviceToHost));

    for (int i = 0; i < rawData.size(); i++) {
        std::vector<float> entry;
        // Put tag results into vector
        for (int j = 0; j < 88; j++) {
            entry.push_back((float) getBit(parsedResults, i*88 + j));
            //printf("entry[-1]);
        }
        // Add abv and ibu
        entry.push_back(atof(collectedData[i][1].c_str()));
        entry.push_back(atof(collectedData[i][2].c_str()));
        results[collectedData[i][0]] = entry;
    }
    
    CudaSafeCall(hipFree(d_descs));
    CudaSafeCall(hipFree(d_results));
    free(parsedResults);
    free(descData);
    free(tags);
    return results;
}
