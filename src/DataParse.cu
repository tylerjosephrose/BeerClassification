#include "hip/hip_runtime.h"
#include <fstream>
#include <map>
#include <stdio.h>
#include <string>
#include <vector>

__constant__ char* c_tags;

/*int getBit(unsigned char *bytes, int bit) {
    return ((bytes[(bit/8)] >> (bit % 8)) & 1);
}

__device__ void setBit(unsigned char *bytes, int bit, int val) {
    if (val == 1)
        bytes[(bit/8)] |= (1 << (bit % 8));
    else
        bytes [(bit/8)] &= ~(1 << (bit % 8));
}

void printBits(unsigned char *ptr, int sizeInBytes) {
    for (int i = 0; i < sizeInBytes * 8; i++) {
        printf("%d", getBit(ptr, i));
    }
    printf("\n");
}*/

__global__ void description_to_tags(char **d_descs, unsigned char *d_results, int sizeEntries, char **d_tags, int sizeTags) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    /*if (idx < sizeTags) {
        //c_tags[idx] = d_tags[idx];
        printf("Thread %d: %s\n", idx, d_tags[idx]);
    }*/
    if (idx >= sizeEntries)
        return;
    //char* desc = d_descs[idx];
    printf("Here with thread for idx: %d, %s\n", idx, d_descs[0]);
    //for (char *tag = *d_tags; tag; tag = *++d_tags) {
    for (int i = 0; i < sizeTags; i++) {
        /*char* tag = c_tags[i];
        //Check if the description contains the tag
        printf("%d: %s\n", idx, tag);
        while (*desc) {
            
        }*/
        printf("Thread %d: %s\n", idx, d_tags[i]);
    }
}

std::map<std::string, std::vector<float> > dataConversion(std::map<std::string, std::vector<std::string> > rawData, std::vector<std::string> tags_internal) {
    //const char** descData = new const char*[2000];
    const char** descData = (const char**) malloc(sizeof(char)*2000*rawData.size());
    int i = 0;
    // Put desc data into char**
    for (std::map<std::string, std::vector<std::string> >::iterator it = rawData.begin(); it != rawData.end(); it++) {
        /*printf("%s: ", it->first.c_str());
        for (uint i = 0; i < it->second.size(); i++) {
            printf("%s - ", it->second[i].c_str());
        }
        printf("\n");*/
        descData[i] = it->second[1].c_str();
        i++;
    }

    // Get Tag data
    const char* tags = (char*) malloc(sizeof(char) * 20 * tags_internal.size());
    memset(tags, '\0', 20*tags_internal.size());
    int i = 0;
    for (std::set<std::string>::iterator it = tags_internal.begin(); it != tags_internal.end(); ++it) {
        for (int j = 0; j < 20; j++) {
            tags[i*20+j] = (*it)[j];
        }
        i++;
    }

    std::map<std::string, std::vector<float> > results;
    
    /* Since we have 11Gb of memory on my GPU we don't need to worry about memory...at 
    85*20 bytes for the tags, 2000 bytes per beer for description, 11 bytes per beer for results
    it would take around 5.5 million beers to run out of memory...We don't have that*/
    const dim3 blockSize(1024, 1, 1);
    const dim3 gridSize(ceil(rawData.size()/1024.0), 1, 1);
    
    char **d_descs, **d_tags;
    unsigned char *d_results, *parsedResults;
    
    hipMalloc(&d_results, 11*rawData.size());
    /*hipMalloc(&d_tags, tags_internal.size()*20);
    hipMemcpy(d_tags, &tags, tags_internal.size()*20, hipMemcpyHostToDevice);
    //hipMemcpyToSymbol(HIP_SYMBOL(c_tags), &tags, tags_internal.size()*20, 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("c_tags"), d_tags, tags_internal.size()*20, 0, hipMemcpyHostToDevice);
    hipMemcpy(d_descs, descData, rawData.size()*2000, hipMemcpyHostToDevice);
    hipMemcpy(d_results, 0, 11*rawData.size(), hipMemcpyHostToDevice);*/

    // Copy descs to device
    hipMalloc(&d_descs, rawData.size()*sizeof(char*));
    char **d_temp_desc;
    d_temp_desc = (char **)malloc(rawData.size()*sizeof(char *));
    for (int i = 0; i < rawData.size(); i++){
        hipMalloc(&(d_temp_desc[i]), 2000*sizeof(char));
        hipMemcpy(d_temp_desc[i], descData[i], 2000*sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_descs+i, &(d_temp_desc[i]), sizeof(char *), hipMemcpyHostToDevice);
    }
    free(d_temp_desc);
    
    // Copy tags to global memory
    hipMalloc(&d_tags, tags_internal.size()*sizeof(char*));
    char **d_temp_tags;
    d_temp_tags = (char **)malloc(tags_internal.size()*sizeof(char*));
    for (int i = 0; i < tags_internal.size(); i++) {
        hipMalloc(&(d_temp_tags[i]), 20*sizeof(char));
        hipMemcpy(d_temp_tags[i], tags[i], 20*sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_tags+i, &(d_temp_tags[i]), sizeof(char *), hipMemcpyHostToDevice);
    }
    free(d_temp_tags);
    
    // Copy tags to constant memory
    
    hipMemcpyToSymbol(HIP_SYMBOL(c_tags), tags, tags_internal.size()*20, 0, hipMemcpyHostToDevice);

    description_to_tags<<<gridSize, blockSize>>>(d_descs, d_results, rawData.size(), d_tags, tags_internal.size());//, d_results, rawData.size(), d_tags);
    
    //parsedResults = (unsigned char*) malloc(11*rawData.size());
    //hipMemcpy(parsedResults, d_results, 11*rawData.size(), hipMemcpyDeviceToHost);
    //delete descData;
    ////hipFree(c_tags);
    hipFree(d_descs);
    hipFree(d_results);
    free(parsedResults);
    free(descData);
    free(tags);
    return results;
}
