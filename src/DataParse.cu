#include "hip/hip_runtime.h"
#include <map>
#include <stdio.h>
#include <string>
#include <vector>

__constant__ char** c_tags;

int getBit(unsigned char *bytes, int bit) {
    return ((bytes[(bit/8)] >> (bit % 8)) & 1);
}

__device__ void setBit(unsigned char *bytes, int bit, int val) {
    if (val == 1)
        bytes[(bit/8)] |= (1 << (bit % 8));
    else
        bytes [(bit/8)] &= ~(1 << (bit % 8));
}

void printBits(unsigned char *ptr, int sizeInBytes) {
    for (int i = 0; i < sizeInBytes * 8; i++) {
        printf("%d", getBit(ptr, i));
    }
    printf("\n");
}

__global__ void description_to_tags(char **d_descs, unsigned char *d_results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    char* desc = d_descs[idx];
    

    //for (char *tag = *c_tags; tag; tag = *++c_tags) {
    for (int i = 0; i < 85; i++) {
        char* tag = c_tags[i];
        //Check if the description contains the tag
        printf("%d: %s\n", idx, tag);
        /*while (*desc) {
            
        }*/
    }
}

std::map<std::string, std::vector<float> > dataConversion(std::map<std::string, std::vector<std::string> > rawData) {
    const char** descData = new const char*[2000];
    int i = 0;
    for (std::map<std::string, std::vector<std::string> >::iterator it = rawData.begin(); it != rawData.end(); it++) {
        /*printf("%s: ", it->first.c_str());
        for (uint i = 0; i < it->second.size(); i++) {
            printf("%s - ", it->second[i].c_str());
        }
        printf("\n");*/
        descData[i] = it->second[1].c_str();
        i++;
    }

    for (int i = 0; i < rawData.size(); i++) {
        printf("%d: %s\n", i, descData[i]);
    }

    // Convert description from string to char**

    std::map<std::string, std::vector<float> > results;
    
    /* Since we have 11Gb of memory on my GPU we don't need to worry about memory...at 
    85*20 bytes for the tags, 2000 bytes per beer for description, 11 bytes per beer for results
    it would take around 5.5 million beers to run out of memory...We don't have that*/
    const dim3 blockSize(1024, 1, 1);
    const dim3 gridSize(ceil(rawData.size()/1024.0, 1, 1);
    
    char **d_descs;
    unsigned char *d_results, *parsedResults;
    
    hipMalloc(&d_descs, rawData.size()*2000);
    hipMalloc(&d_results, 11*rawData.size());
    cudaMemcpytoSymbol(c_tags, tags, tags.size()*20);
    hipMemcpy(d_descs, descData, rawData.size()*2000, hipMemcpyHostToDevice);
    hipMemcpy(d_results, 0, 11*rawData.size(), hipMemcpyHostToDevice);
    
    description_to_tags<<<gridSize, blockSize>>>(d_descs, d_results);
    
    parsedResults = malloc(11*rawData.size());
    hipMemcpy(parsedResults, d_results, 11*rawData.size(), hipMemcpyDeviceToHost);
    delete descData;
    hipFree(d_tags);
    hipFree(d_descs);
    hipFree(d_results);
    
    return results;
}
