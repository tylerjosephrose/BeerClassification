#include "hip/hip_runtime.h"
#include <fstream>
#include <map>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <vector>

#include "BeerEntry.h"

#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

int getBit(unsigned char *bytes, int bit) {
    return ((bytes[(bit/8)] >> (bit % 8)) & 1);
}

__global__ void description_to_tags_optimized(char **d_descs, unsigned char *d_results, int sizeEntries, char **d_tags, int sizeTags) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int descNum = idx/sizeTags;
    int tagNum = idx%sizeTags;
    __shared__ char* tags[85];
    if (threadIdx.x < sizeTags) {
        char tag[20];
        for (int i = 0; i < 20; i++) {
            tag[i] = d_tags[threadIdx.x][i];
        }
        tags[threadIdx.x] = tag;
    }

    __syncthreads();

    if (idx >= (sizeEntries * sizeTags))
        return;

    int i = 0;
    int match = 0;
    char descLetter = d_descs[descNum][i];
    char tagLetter = tags[tagNum][match];
    while (descLetter != '\0') {
        // Convert desc letter to lowercase for matching
        if ('A'<=descLetter && descLetter<='Z'){
            descLetter=char(((int)descLetter)+32);
        }
        if (descLetter == tagLetter) {
            match++;
            tagLetter = tags[tagNum][match];
            if (tagLetter == '\0') {
                // We have a match so put it in the results
                switch (tagNum % 8) {
                    case 0:
                        d_results[descNum] |= 0b10000000;
                        break;
                    case 1:
                        d_results[descNum] |= 0b01000000;
                        break;
                    case 2:
                        d_results[descNum] |= 0b00100000;
                        break;
                    case 3:
                        d_results[descNum] |= 0b00010000;
                        break;
                    case 4:
                        d_results[descNum] |= 0b00001000;
                        break;
                    case 5:
                        d_results[descNum] |= 0b00000100;
                        break;
                    case 6:
                        d_results[descNum] |= 0b00000010;
                        break;
                    case 7:
                        d_results[descNum] |= 0b00000001;
                        break;
                }
            }
        } else {
            match = 0;
            tagLetter = tags[tagNum][match];
        }
        i++;
        descLetter = d_descs[descNum][i];
    }
}

__global__ void description_to_tags_original(char **d_descs, unsigned char *d_results, int sizeEntries, char **d_tags, int sizeTags) {
    /*if (threadIdx.x > 0) 
        return;
    for (int num = 0; num < sizeTags; num++) {*/
    int num = threadIdx.x;
    for (int i = 0; i < sizeEntries; i++) {
        int spacing = sizeEntries/sizeTags;
        int idx = (i + threadIdx.x*spacing) % sizeEntries;

        // Copy the desc locally so we don't have read conflicts
        int j = 0;
        char desc[2000];
        while (d_descs[idx][j] != '\0') {
            desc[j] = d_descs[idx][j];
            j++;
        }
        desc[j] = '\0';

        //char* tag = d_tags[threadIdx.x];
        char* tag = d_tags[num];
        int tagLength = 0;
        while (tag[tagLength] != '\0')
            tagLength++;
        
        int match = 0;
        bool positiveMatch = false;
        j = 0;
        while (desc[j] != '\0') {
            char descLetter = desc[j];
            if ('A'<=descLetter && descLetter<='Z'){
                descLetter=char(((int)descLetter)+32);
            }

            if (descLetter == tag[match]) {
                match++;
                if (match == tagLength) {
                    positiveMatch = true;
                    break;
                }
            }
            else
                match = 0;
            j++;
        }
        
        if (positiveMatch) {
            //printf("Thread %d:%d is looking for %s\t%s\n", threadIdx.x, idx, tag, "True!");
            // Since 11 bytes are given for each entry we need to find the byte that we are in and add the specific flag we need
            switch (num % 8) {
                case 0:
                    d_results[(num/8) * i] |= 0b10000000;
                    break;
                case 1:
                    d_results[(num/8) * i] |= 0b01000000;
                    break;
                case 2:
                    d_results[(num/8) * i] |= 0b00100000;
                    break;
                case 3:
                    d_results[(num/8) * i] |= 0b00010000;
                    break;
                case 4:
                    d_results[(num/8) * i] |= 0b00001000;
                    break;
                case 5:
                    d_results[(num/8) * i] |= 0b00000100;
                    break;
                case 6:
                    d_results[(num/8) * i] |= 0b00000010;
                    break;
                case 7:
                    d_results[(num/8) * i] |= 0b00000001;
                    break;
            }
        }
        else {
            //printf("Thread %d:%d is looking for %s\t%s\n", threadIdx.x, idx, tag, "False");
            switch (num % 8) {
                case 0:
                    d_results[(num/8) * i] &= ~0b10000000;
                    break;
                case 1:
                    d_results[(num/8) * i] &= ~0b01000000;
                    break;
                case 2:
                    d_results[(num/8) * i] &= ~0b00100000;
                    break;
                case 3:
                    d_results[(num/8) * i] &= ~0b00010000;
                    break;
                case 4:
                    d_results[(num/8) * i] &= ~0b00001000;
                    break;
                case 5:
                    d_results[(num/8) * i] &= ~0b00000100;
                    break;
                case 6:
                    d_results[(num/8) * i] &= ~0b00000010;
                    break;
                case 7:
                    d_results[(num/8) * i] &= ~0b00000001;
                    break;
            }
        }
        __syncthreads();
    }
    //}
}

std::vector<BeerEntry> dataConversion(std::map<std::string, std::vector<std::string> > rawData, std::vector<std::string> tags_internal) {
    const char** descData = (const char**) malloc(sizeof(char)*2000*rawData.size());
    // Collected data is other data that we need to return we collect it so we don't need to iterate through the map again and worry about order.
    // The data is the style of the beer, abv, ibu
    std::vector< std::vector<std::string> > collectedData;
    unsigned int i = 0;
    // Put desc data into char**
    for (std::map<std::string, std::vector<std::string> >::iterator it = rawData.begin(); it != rawData.end(); it++) {
        descData[i] = it->second[1].c_str();
        std::vector<std::string> data;
        data.push_back(it->second[4].c_str());
        data.push_back(it->second[2].c_str());
        data.push_back(it->second[3].c_str());
        collectedData.push_back(data);
        i++;
    }

    // Get Tag data
    const char** tags = (const char**) malloc(sizeof(char) * 20 * tags_internal.size());
    for (i = 0; i < tags_internal.size(); i++) {
        tags[i] = tags_internal[i].c_str();
    }

    std::vector<BeerEntry> results;
    
    /* Since we have 11Gb of memory on my GPU we don't need to worry about memory...at 
    85*20 bytes for the tags, 2000 bytes per beer for description, 11 bytes per beer for results
    it would take around 5.5 million beers to run out of memory...We don't have that*/
    //const dim3 blockSize(tags_internal.size(), 1, 1);
    //const dim3 gridSize(1, 1, 1);
    
    char **d_descs, **d_tags;
    unsigned char *d_results;
    
    unsigned char *parsedResults = (unsigned char*) malloc(11*rawData.size());
    size_t size = 11*rawData.size();
    CudaSafeCall(hipMalloc(&d_results, size));

    // Copy descs to device
    CudaSafeCall(hipMalloc(&d_descs, rawData.size()*sizeof(char*)));
    char **d_temp_desc;
    d_temp_desc = (char **)malloc(rawData.size()*sizeof(char *));
    for (unsigned int i = 0; i < rawData.size(); i++){
        CudaSafeCall(hipMalloc(&(d_temp_desc[i]), 2000*sizeof(char)));
        CudaSafeCall(hipMemcpy(d_temp_desc[i], descData[i], 2000*sizeof(char), hipMemcpyHostToDevice));
        CudaSafeCall(hipMemcpy(d_descs+i, &(d_temp_desc[i]), sizeof(char *), hipMemcpyHostToDevice));
    }
    free(d_temp_desc);

    // Copy tags to global memory
    CudaSafeCall(hipMalloc(&d_tags, tags_internal.size()*sizeof(char*)));

    char **d_temp_tags;
    d_temp_tags = (char **)malloc(tags_internal.size()*sizeof(char*));
    for (unsigned int i = 0; i < tags_internal.size(); i++) {
        CudaSafeCall(hipMalloc(&(d_temp_tags[i]), 20*sizeof(char)));
        CudaSafeCall(hipMemcpy(d_temp_tags[i], tags[i], 20*sizeof(char), hipMemcpyHostToDevice));
        CudaSafeCall(hipMemcpy(d_tags+i, &(d_temp_tags[i]), sizeof(char *), hipMemcpyHostToDevice));
    }
    free(d_temp_tags);

    //description_to_tags_original<<<gridSize, blockSize>>>(d_descs, d_results, rawData.size(), d_tags, tags_internal.size());
    const dim3 blockSizeOptimized(1024, 1, 1);
    const dim3 gridSizeOptimized(ceil((rawData.size()*tags_internal.size())/1024.0), 1, 1);
    description_to_tags_optimized<<<gridSizeOptimized, blockSizeOptimized>>>(d_descs, d_results, rawData.size(), d_tags, tags_internal.size());

    CudaSafeCall(hipMemcpy(parsedResults, d_results, 11*rawData.size(), hipMemcpyDeviceToHost));

    for (unsigned int i = 0; i < rawData.size(); i++) {
        BeerEntry entry;
        std::vector<float> values;
        // Put tag results into vector
        for (int j = 0; j < 88; j++) {
            values.push_back((float) getBit(parsedResults, i*88 + j));
        }
        // Add abv and ibu
        values.push_back(atof(collectedData[i][1].c_str()));
        values.push_back(atof(collectedData[i][2].c_str()));
        entry.values = values;
        entry.style = collectedData[i][0];
        results.push_back(entry);
    }
    
    CudaSafeCall(hipFree(d_descs));
    CudaSafeCall(hipFree(d_results));
    free(parsedResults);
    free(descData);
    free(tags);
    return results;
}
