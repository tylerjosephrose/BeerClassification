
#include <hip/hip_runtime.h>
#include <map>
#include <stdio.h>
#include <string>
#include <vector>

void descriptionToTags(std::map<std::string, std::vector<std::string> > rawData) {
    // make kernel call
    printf("in cuda code size: %d\n", rawData.size());
    for (std::map<std::string, std::vector<std::string> >::iterator it = rawData.begin(); it != rawData.end(); it++) {
        printf("%s: ", it->first.c_str());
        for (uint i = 0; i < it->second.size(); i++) {
            //printf("%s ", it->second[i].c_str());
        }
        printf("\n");
    }
}